#include "hip/hip_runtime.h"
/* Kintsakis Athanasios AEM */

#define inf 99999

__global__ void funct1(int n, int k, float* x, int* qx)
{

	__shared__ float dBlock[1024];
	__shared__ float QBlock[1024];
	int i=(threadIdx.x>>5);
	int j=threadIdx.x&31;

	int index1=(k*32+i)*n+k*32+j;
	dBlock[threadIdx.x]=x[index1];
	QBlock[threadIdx.x]=qx[index1];
	

	int k1=k*32;
	int l;

	for(l=0;l<32;l++)
	{
		
		syncthreads();
		float temp2=dBlock[(i<<5)+l]+dBlock[(l<<5)+j];

		if(dBlock[threadIdx.x]>temp2)
		{

			dBlock[threadIdx.x]=temp2;
			QBlock[threadIdx.x]=l+k1;
		}

	}

	x[index1]=dBlock[threadIdx.x];
	qx[index1]=QBlock[threadIdx.x];
	

}

__global__ void funct2(int n, int k, float* x, int* qx)
{
if(blockIdx.y==0)
{
	int i=(threadIdx.x>>5);
	int j=threadIdx.x&31;
	int k1=k*32;

	
	__shared__ float dBlock[1024];
	__shared__ float QcBlock[1024];
	__shared__ float cBlock[1024];

	dBlock[threadIdx.x]=x[(k1+i)*n+k1+j];

	int l;

	int add=0;

	if(blockIdx.x>=k) //auto einai gia na pidikseis to kentriko block
	{
		add=1;
		
	}

	int index1=(k1+i)*n+(blockIdx.x+add)*32+j;

	 cBlock[threadIdx.x]= x[index1];
	QcBlock[threadIdx.x]=qx[index1];
			
	
		for(l=0;l<32;l++)
		{
			syncthreads();
			
			float temp2=dBlock[i*32+l]+cBlock[l*32+j];

			if(cBlock[threadIdx.x]>temp2)
			{
				cBlock[threadIdx.x]=temp2;

				QcBlock[threadIdx.x]=l+k1;
			}

		}
		x[index1]=cBlock[threadIdx.x];
		qx[index1]=QcBlock[threadIdx.x];
}
else
{
	int i=(threadIdx.x>>5);
	int j=threadIdx.x&31;
	int k1=k*32;
	
	__shared__ float dBlock[1024];
	__shared__ float QcBlock[1024];
	__shared__ float cBlock[1024];

	dBlock[threadIdx.x]=x[(k1+i)*n+k1+j];

	int l;

	int add=0;

	if(blockIdx.x>=k) //auto einai gia na pidikseis to kentriko block
	{
		add=1;
		
	}

		int index1=((blockIdx.x+add)*32+i)*n+k1+j;
		cBlock[threadIdx.x]=x[index1];
		QcBlock[threadIdx.x]=qx[index1];
		
		
		for(l=0;l<32;l++)
		{
			syncthreads();

			float temp2=cBlock[i*32+l]+dBlock[l*32+j];

			if(cBlock[threadIdx.x]>temp2)
			{
				cBlock[threadIdx.x]=temp2;

				QcBlock[threadIdx.x]=l+k1;
			}

		}

	x[index1]=cBlock[threadIdx.x];
	qx[index1]=QcBlock[threadIdx.x];

}

}

__global__ void funct3(int n, int k, float* x, int* qx)
{
	int i=(threadIdx.x>>5);
	int j=threadIdx.x&31;
	int k1=k*32;
	int addx=0;
	int addy=0;

	__shared__ float dyBlock[1024];
	__shared__ float dxBlock[1024];
	__shared__ float QcBlock[1024];
	__shared__ float cBlock[1024];

	if(blockIdx.x>=k)
	{
		addx=1;
		
	}
	if(blockIdx.y>=k)
	{
		addy=1;
		
	}	

	dxBlock[threadIdx.x]=x[ ((k<<5)+i)*n + ((blockIdx.y+addy)<<5)+j];
	dyBlock[threadIdx.x]=x[ (((blockIdx.x+addx)<<5) +i)*n+(k<<5)+j];
	int index1=(((blockIdx.x+addx)<<5) +i)*n + ((blockIdx.y+addy)<<5) + j;

	
	cBlock[threadIdx.x]=  x[index1];
	QcBlock[threadIdx.x]=qx[index1];
	int l;
	for(l=0;l<32;l++)
		{
			syncthreads();

			float temp2=dyBlock[i*32+l]+dxBlock[l*32+j];

			if(cBlock[threadIdx.x]>temp2)
			{
				cBlock[threadIdx.x]=temp2;

				QcBlock[threadIdx.x]=l+k1;
			}

		}


 	x[index1]=cBlock[threadIdx.x];
	qx[index1]=QcBlock[threadIdx.x];
}

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include  <time.h>
#include <sys/time.h>


int main(int argc, char **argv)
{

  struct timeval first, second, lapsed,third,fourth,lapsed2;
  struct timezone tzp,tzp2;
	float *host_A, *host_D;
	int *host_Q;
	float *dev_x;
	int *dev_qx;
	float *A;
	int *Q;
	float *D;

	int i,j;
	int k=0;
	float tolerance=0.001;
	int n = atoi(argv[1]);
	
	printf("\n");
	printf("RUNNING WITH %d VERTICES \n",n);
	printf("\n");

	hipMalloc(&dev_x, n*n*sizeof(float));
	hipMalloc(&dev_qx, n*n*sizeof(float));
	

    //CPU arrays
	A=(float *)malloc(n*n*sizeof(float));  //arxikos pinakas A
	D=(float *)malloc(n*n*sizeof(float));  //arxikos pinakas D
	Q=(int *)malloc(n*n*sizeof(int));  //arxikos pinakas Q
	
	//GPU arrays
	host_A=(float *)malloc(n*n*sizeof(float));
	host_D=(float *)malloc(n*n*sizeof(float));
	host_Q=(int *)malloc(n*n*sizeof(int));
	


	
	srand(time(NULL));

	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		{
			Q[i*n+j]=-1;
		}
	}



	
	for(i=0;i<n;i++)
	{
		
		for(j=0;j<n;j++)
		{
			if(i==j)
			{A[i*n+j]=0;}
			else
			{
				A[i*n+j]=1200 * (float) rand()/RAND_MAX + 1;
				//A[i*n+j]=i*n+j;//(int)rand()%10000;
				if(A[i*n+j]>1000)
				{
					A[i*n+j]=inf;
					Q[i*n+j]=-2;
				}
			}
		}
	}


/*	A[0][0]=0;
	A[0][1]=inf;
	A[0][2]=1;

	A[1][0]=2;
	A[1][1]=0;
	A[1][2]=10;

	A[2][0]=inf;
	A[2][1]=4;
	A[2][2]=0;
	*/
	for(i=0;i<n;i++)    
	{
		for(j=0;j<n;j++)
		{
			D[i*n+j]=A[i*n+j];
		}

	}

	
	for(i=0;i<n;i++)    //Antigrafi tou A ston host_A
	{
		for(j=0;j<n;j++)
		{
			host_A[i*n+j]=A[i*n+j];
		}

	}


	for(i=0;i<n;i++)    //Antigrafi tou Q ston host_Q
	{
		for(j=0;j<n;j++)
		{
			host_Q[i*n+j]=Q[i*n+j];
		}

	}
gettimeofday(&third, &tzp2);
////////////////////////////First Mem Copy////////////////////

 	gettimeofday(&first, &tzp);
	hipMemcpy(dev_x, host_A, n*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_qx, host_Q, n*n*sizeof(int), hipMemcpyHostToDevice);
	


gettimeofday(&second, &tzp);

  if(first.tv_usec>second.tv_usec){
    second.tv_usec += 1000000;
    second.tv_sec--;
  }
  
  lapsed.tv_usec = second.tv_usec - first.tv_usec;
  lapsed.tv_sec = second.tv_sec - first.tv_sec;

  printf("First Transfer CPU to GPU  Time elapsed: %lu,%06lu s\n", lapsed.tv_sec, lapsed.tv_usec);
////////////////////////////////////////////////////GPU Calculation////////////////////////////////
		
	gettimeofday(&first, &tzp);	

	dim3 bk2(n/32-1,2);
	dim3 bk3(n/32-1,n/32-1);
	int gputhreads=1024;
	for(k=0;k<n/32;k++)
	{	
		
		funct1<<<1,gputhreads>>>(n,k,dev_x,dev_qx);

		funct2<<<bk2,gputhreads>>>(n,k,dev_x,dev_qx);

		funct3<<<bk3,gputhreads>>>(n,k,dev_x,dev_qx);

	}
	
hipDeviceSynchronize();

gettimeofday(&second, &tzp);

  if(first.tv_usec>second.tv_usec){
    second.tv_usec += 1000000;
    second.tv_sec--;
  }
  
  lapsed.tv_usec = second.tv_usec - first.tv_usec;
  lapsed.tv_sec = second.tv_sec - first.tv_sec;

  printf("GPU Calculation Time elapsed: %lu,%06lu s\n", lapsed.tv_sec, lapsed.tv_usec);
  printf("\n");

//////////////////////////////////////////////////////////////////////////Second Mem Copy////////////////////
 gettimeofday(&first, &tzp);

hipMemcpy(host_D, dev_x, n*n*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(host_Q, dev_qx, n*n*sizeof(int), hipMemcpyDeviceToHost);
gettimeofday(&second, &tzp);

  if(first.tv_usec>second.tv_usec){
    second.tv_usec += 1000000;
    second.tv_sec--;
  }
  
  lapsed.tv_usec = second.tv_usec - first.tv_usec;
  lapsed.tv_sec = second.tv_sec - first.tv_sec;

  printf("Second Transfer GPU to CPU  Time elapsed: %lu,%06lu s\n", lapsed.tv_sec, lapsed.tv_usec);
  printf("\n");
//////////////////////////////////////////////////////////////////////

gettimeofday(&fourth, &tzp2); //total time

  if(third.tv_usec>fourth.tv_usec){
    fourth.tv_usec += 1000000;
    fourth.tv_sec--;
  }
  
  lapsed2.tv_usec = fourth.tv_usec - third.tv_usec;
  lapsed2.tv_sec = fourth.tv_sec - third.tv_sec;

  printf("TOTAL GPU + TRANSFERS  Time elapsed: %lu,%06lu s\n", lapsed2.tv_sec, lapsed2.tv_usec);


//////////////////////////////////////////////////////////////


//CPU RUN 
printf(" THIS IS CPU \n");
printf("\n");
 gettimeofday(&first, &tzp);

	for(k=0;k<n;k++)
	{
		for(i=0;i<n;i++)
		{
			for(j=0;j<n;j++)
			{

				if((D[i*n+k]+D[k*n+j])<D[i*n+j])
				{
					D[i*n+j]=D[i*n+k]+D[k*n+j];
					Q[i*n+j]=k;					
				}
				if(D[i*n+j]==inf)
				{
					//Q[i*n+j]=-2;
				}
			}
		}
	}
/////////////////////////////////////////////////////////////////
gettimeofday(&second, &tzp);

  if(first.tv_usec>second.tv_usec){
    second.tv_usec += 1000000;
    second.tv_sec--;
  }
  
  lapsed.tv_usec = second.tv_usec - first.tv_usec;
  lapsed.tv_sec = second.tv_sec - first.tv_sec;

  printf("CPU Time elapsed: %lu,%06lu s\n", lapsed.tv_sec, lapsed.tv_usec);
/////////////////////////////////////////////////////

printf(" \n");

		
printf(" \n");

/////////////FROM HERE AND UNDER ARE VALIDATION RUNS

printf("VALIDATING THAT D array from CPU and host_D array from GPU match... \n");


for(i=0;i<n;i++)
{
	for(j=0;j<n;j++)
	{
		if(abs(D[i*n+j]-host_D[i*n+j])>tolerance)
		{
			
			printf("ERROR MISMATCH in array D i %d j %d CPU SAYS %f and GPU SAYS %f \n",i,j,D[i*n+j],host_D[i*n+j]);
		}
	}
}
printf("OK \n");


for(i=0;i<n;i++)
{
	for(j=0;j<n;j++)
	{
		if(Q[i*n+j]!=host_Q[i*n+j])
		{
			//printf("ERROR MISMATCH in array Q i %d j %d, CPU IS %d, GPU IS %d \n",i,j,Q[i*n+j],host_Q[i*n+j]);
			//this is disabled as the two pathways are not equal, but the distance is equal
		}
	}
}
printf("OK \n");
//printf("more validating .................... Patience!....\n");

printf("Initiating Path Rebuilding Check for CPU DATA... \n");
for(i=0;i<n;i++)
{
	for(j=0;j<n;j++)
	{

		if( (i==j) && (D[i*n+j]!=0) )
		{
			printf(" ERROR, i=j but element is not 0 \n");
		}

		if(D[i*n+j]==inf)
		{
			if(Q[i*n+j]!=-2)
			{
				printf("ERROR, No path exists between i %d and %d but array Q saysotherwise \n",i,j);
			}
			
		}
		else if(Q[i*n+j]==-1)
		{
			if(abs(A[i*n+j]-D[i*n+j])>tolerance)
			{
				printf("ERROR, from i %d to j %d cheapest route should be the direct one. D array not equal to array A cost \n",i,j);
			}
		}
		else
		{
			float temppath=0;
			int tempi=i;
			int tempj=j;
		
			while(Q[tempi*n+tempj]>=-1)
			{
				if(Q[tempi*n+tempj]==-1)
				{
					temppath=temppath+A[tempi*n+tempj];
					break;
				}
				temppath=temppath+D[tempi*n+Q[tempi*n+tempj]];
				tempi=Q[tempi*n+tempj];
				
			}

			if(abs(temppath-D[i*n+j])>tolerance)
			{
				printf("ERROR 2 in i %d j %d, expected %f and is %f \n",i,j,temppath,D[i*n+j]);
			}
			/*else
			{
		printf("CPU PART : ||| COMPUTED ARRAY D SAYS %f AND TEMPPATH CALCULATED IS %f \n",D[i*n+j],temppath);
			}*/
		}
			
				
	}
}
printf("OK \n");
printf("Initiating Path Rebuilding Check for GPU DATA.... \n");

for(i=0;i<n;i++)
{
	for(j=0;j<n;j++)
	{

		if( (i==j) && (host_D[i*n+j]!=0) )
		{
			printf(" ERROR, i=j but element is not 0 \n");
		}

		if(host_D[i*n+j]==inf)
		{
			if(host_Q[i*n+j]!=-2)
			{
				printf("ERROR, No path exists between i %d and %d but array Q saysotherwise \n",i,j);
			}
			
		}
		else if(host_Q[i*n+j]==-1)
		{
			if(abs(host_A[i*n+j]-host_D[i*n+j])>tolerance)
			{
				printf("ERROR, from i %d to j %d cheapest route should be the direct one. host_D array not equal to host_A cost \n",i,j);
			}
		}
		else
		{
			float temppath=0;
			int tempi=i;
			int tempj=j;
		
			while(host_Q[tempi*n+tempj]>=-1)
			{
				if(host_Q[tempi*n+tempj]==-1)
				{
					temppath=temppath+host_A[tempi*n+tempj];
					break;
				}
				temppath=temppath+host_D[tempi*n+host_Q[tempi*n+tempj]];
				tempi=host_Q[tempi*n+tempj];
				
			}
				

			if(abs(temppath-host_D[i*n+j])>tolerance)
			{
				printf("Path REBUILT ERROR in cheching dinstance from in i %d j %d, expected %f and we found %f \n",i,j,temppath,host_D[i*n+j]);
			}
			/*else
			{
		printf("GPU PART : COMPUTED ARRAY HOSTD SAYS %f, ARRAY FROM CPU D SAYS %f AND TEMPPATH CALCULATED IS %f \n",D[i*n+j],host_D[i*n+j],temppath);
			}*/
		}
			
				
	}
}
printf("OK \n");


printf("Head to head of shortest path between any two nodes based on array Q from CPU and array Q from blocked GPU algorithm.... \n");

for(i=0;i<n;i++)
{
	for(j=0;j<n;j++)
	{
			//GPU
			float gputemppath=0;
			int gputempi=i;
			int gputempj=j;

			while(host_Q[gputempi*n+gputempj]>=-1)
			{
				if(host_Q[gputempi*n+gputempj]==-1)
				{
					gputemppath=gputemppath+host_D[gputempi*n+gputempj];
					break;
				}
				gputemppath=gputemppath+host_D[gputempi*n+host_Q[gputempi*n+gputempj]];
				gputempi=host_Q[gputempi*n+gputempj];
				
			}
			//GPU

			//CPU
			float cputemppath=0;
			int cputempi=i;
			int cputempj=j;
		
			while(Q[cputempi*n+cputempj]>=-1)
			{
				if(Q[cputempi*n+cputempj]==-1)
				{
					cputemppath=cputemppath+A[cputempi*n+cputempj];
					break;
				}
				cputemppath=cputemppath+D[cputempi*n+Q[cputempi*n+cputempj]];
				cputempi=Q[cputempi*n+cputempj];
				
			}
			//CPU
			if(abs(gputemppath-cputemppath)>tolerance)
			{
				printf("ERROR mismatch gpupath is %f cpupath is %f \n",gputemppath,cputemppath);
			}
			else
			{
				//printf("WAS OK, VALUES, CPU %f, GPU %f \n",cputemppath,gputemppath);
			}
	}


}
printf("ALL OK WE ARE DONE \n");
return 0;

}

