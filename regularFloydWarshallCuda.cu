#include "hip/hip_runtime.h"
/**
#Copyright 2013 Athanassios Kintsakis

#Licensed under the Apache License, Version 2.0 (the "License");
#you may not use this file except in compliance with the License.
#You may obtain a copy of the License at

    #http://www.apache.org/licenses/LICENSE-2.0

#Unless required by applicable law or agreed to in writing, software
#distributed under the License is distributed on an "AS IS" BASIS,
#WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
#See the License for the specific language governing permissions and
#limitations under the License.

# Author: Athanassios Kintsakis
# contact: akintsakis@issel.ee.auth.gr, athanassios.kintsakis@gmail.com
 **/
#define inf 9999

__global__ void funct(int n, int k, float* x, int* qx) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int j = ix & (n - 1);
    float temp2 = x[ix - j + k] + x[k * n + j];
    if (x[ix] > temp2) {
        x[ix] = temp2;
        qx[ix] = k;
    }
    if (x[ix] == inf) {
        qx[ix] = -2;
    }
}

__global__ void funct2(int n, int k, float* x, int* qx) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int j = ix & (n - 1);
    float temp2 = x[ix - j + k] + x[k * n + j];
    if (x[ix] > temp2) {
        x[ix] = temp2;
        qx[ix] = k;
    }
}


#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include  <time.h>
#include <sys/time.h>

int main(int argc, char **argv) {

    struct timeval first, second, lapsed, third, fourth, lapsed2;
    struct timezone tzp, tzp2;
    float *host_A, *host_D;
    int *host_Q;
    float *dev_x;
    int *dev_qx;
    float *A;
    int *Q;
    float *D;
    float tolerance = 0.001;

    int i, j, bk;
    int k = 0;
    int n = atoi(argv[1]);

    printf("\n");
    printf("RUNNING WITH %d VERTICES \n", n);
    printf("\n");

    hipMalloc(&dev_x, n * n * sizeof (float));
    hipMalloc(&dev_qx, n * n * sizeof (float));

    //CPU arrays
    A = (float *) malloc(n * n * sizeof (float)); //arxikos pinakas A
    D = (float *) malloc(n * n * sizeof (float)); //arxikos pinakas D
    Q = (int *) malloc(n * n * sizeof (int)); //arxikos pinakas Q

    //GPU arrays
    host_A = (float *) malloc(n * n * sizeof (float));
    host_D = (float *) malloc(n * n * sizeof (float));
    host_Q = (int *) malloc(n * n * sizeof (int));

    srand(time(NULL));
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            if (i == j) {
                A[i * n + j] = 0;
            } else {
                A[i * n + j] = 1200 * (float) rand() / RAND_MAX + 1;
                if (A[i * n + j] > 1000) {
                    A[i * n + j] = inf;
                }
            }
        }
    }
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            Q[i * n + j] = -1;
        }
    }
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            D[i * n + j] = A[i * n + j];
        }

    }
    for (i = 0; i < n; i++) //Antigrafi tou A ston host_A
    {
        for (j = 0; j < n; j++) {
            host_A[i * n + j] = A[i * n + j];
        }

    }
    for (i = 0; i < n; i++) //Antigrafi tou Q ston host_Q
    {
        for (j = 0; j < n; j++) {
            host_Q[i * n + j] = Q[i * n + j];
        }
    }
    gettimeofday(&third, &tzp2);
    ////////////////////////////First Mem Copy////////////////////
    gettimeofday(&first, &tzp);
    hipMemcpy(dev_x, host_A, n * n * sizeof (float), hipMemcpyHostToDevice);
    hipMemcpy(dev_qx, host_Q, n * n * sizeof (int), hipMemcpyHostToDevice);
    gettimeofday(&second, &tzp);
    if (first.tv_usec > second.tv_usec) {
        second.tv_usec += 1000000;
        second.tv_sec--;
    }
    lapsed.tv_usec = second.tv_usec - first.tv_usec;
    lapsed.tv_sec = second.tv_sec - first.tv_sec;
    printf("First Transfer CPU to GPU  Time elapsed: %lu, %lu s\n", lapsed.tv_sec, lapsed.tv_usec);
    ////////////////////////////////////////////////////GPU Calculation////////////////////////////////

    bk = (int) (n * n / 512);
    int gputhreads = 512;
    if (bk > 0) {
        gputhreads = 512;
    } else {
        bk = 1;
        gputhreads = n*n;
    }
    printf(" \n");
    printf("BLOCKS :   %d      GPU THREADS:     %d \n", bk, gputhreads);
    printf(" \n");
    gettimeofday(&first, &tzp);
    funct << <bk, gputhreads>>>(n, k, dev_x, dev_qx);
    for (k = 1; k < n; k++) {
        funct2 << <bk, gputhreads>>>(n, k, dev_x, dev_qx);
    }
    hipDeviceSynchronize();
    gettimeofday(&second, &tzp);
    if (first.tv_usec > second.tv_usec) {
        second.tv_usec += 1000000;
        second.tv_sec--;
    }

    lapsed.tv_usec = second.tv_usec - first.tv_usec;
    lapsed.tv_sec = second.tv_sec - first.tv_sec;
    printf("GPU Calculation Time elapsed: %lu, %lu s\n", lapsed.tv_sec, lapsed.tv_usec);
    printf("\n");

    //////////////////////////////////////////////////////////////////////////Second Mem Copy////////////////////
    gettimeofday(&first, &tzp);
    hipMemcpy(host_D, dev_x, n * n * sizeof (float), hipMemcpyDeviceToHost);
    hipMemcpy(host_Q, dev_qx, n * n * sizeof (int), hipMemcpyDeviceToHost);
    gettimeofday(&second, &tzp);
    if (first.tv_usec > second.tv_usec) {
        second.tv_usec += 1000000;
        second.tv_sec--;
    }

    lapsed.tv_usec = second.tv_usec - first.tv_usec;
    lapsed.tv_sec = second.tv_sec - first.tv_sec;
    printf("Second Transfer GPU to CPU  Time elapsed: %lu, %lu s\n", lapsed.tv_sec, lapsed.tv_usec);
    printf("\n");
    //////////////////////////////////////////////////////////////////////

    gettimeofday(&fourth, &tzp2); //total time
    if (third.tv_usec > fourth.tv_usec) {
        fourth.tv_usec += 1000000;
        fourth.tv_sec--;
    }
    lapsed2.tv_usec = fourth.tv_usec - third.tv_usec;
    lapsed2.tv_sec = fourth.tv_sec - third.tv_sec;
    printf("TOTAL GPU + TRANSFERS  Time elapsed: %lu, %lu s\n", lapsed2.tv_sec, lapsed2.tv_usec);
    //////////////////////////////////////////////////////////////


    //CPU RUN 
    printf("\n");
    printf("\n");
    printf(" Now running on CPU... \n");
    printf("\n");
    gettimeofday(&first, &tzp);
    for (k = 0; k < n; k++) {
        for (i = 0; i < n; i++) {
            for (j = 0; j < n; j++) {

                if ((D[i * n + k] + D[k * n + j]) < D[i * n + j]) {
                    D[i * n + j] = D[i * n + k] + D[k * n + j];
                    Q[i * n + j] = k;
                }
                if (D[i * n + j] == inf) {
                    //Q[i*n+j]=-2;
                }
            }
        }
    }
    /////////////////////////////////////////////////////////////////
    gettimeofday(&second, &tzp);
    if (first.tv_usec > second.tv_usec) {
        second.tv_usec += 1000000;
        second.tv_sec--;
    }
    lapsed.tv_usec = second.tv_usec - first.tv_usec;
    lapsed.tv_sec = second.tv_sec - first.tv_sec;
    printf("CPU Time elapsed: %lu,%06lu s\n", lapsed.tv_sec, lapsed.tv_usec);
    /////////////////////////////////////////////////////
    printf(" \n");
    printf(" \n");
    /////////////FROM HERE AND UNDER ARE VALIDATION RUNS

    printf("VALIDATING THAT D array from CPU and host_D array from GPU match... \n");
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            if (abs(D[i * n + j] - host_D[i * n + j]) > tolerance) {

                printf("ERROR MISMATCH in array D i %d j %d CPU SAYS %f and GPU SAYS %f \n", i, j, D[i * n + j], host_D[i * n + j]);
            }
        }
    }
    printf("OK \n");
    /*
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            if (Q[i * n + j] != host_Q[i * n + j]) {
                //printf("ERROR MISMATCH in array Q i %d j %d, CPU IS %d, GPU IS %d \n",i,j,Q[i*n+j],host_Q[i*n+j]);
                //this is disabled as the two pathways are not equal, but the distance is equal
            }
        }
    }
    printf("OK \n");
     */
    /* Warning! These tests may cause an infinite loop in case a mismatch exists between cpu and gpu. By default they are
     * disabled and only exist for strictly diagnostic purposes.    
     */
    /*
    printf("WARNING now running path rebuilding tests.. these tests may cause an infinite loop in case there is a mismatch between the CPU and GPU calculations\n");
    printf("Initiating Path Rebuilding Check for CPU DATA... \n");
    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            if ((i == j) && (D[i * n + j] != 0)) {
                printf(" ERROR, i=j but element is not 0 \n");
            }
            if (D[i * n + j] == inf) {
                if (Q[i * n + j] != -2) {
                    printf("ERROR, No path exists between i %d and %d but array Q saysotherwise \n", i, j);
                }
            } else if (Q[i * n + j] == -1) {
                if (abs(A[i * n + j] - D[i * n + j]) > tolerance) {
                    printf("ERROR, from i %d to j %d cheapest route should be the direct one. D array not equal to array A cost \n", i, j);
                }
            } else {
                float temppath = 0;
                int tempi = i;
                int tempj = j;
                while (Q[tempi * n + tempj] >= -1) {
                    if (Q[tempi * n + tempj] == -1) {
                        temppath = temppath + A[tempi * n + tempj];
                        break;
                    }
                    temppath = temppath + D[tempi * n + Q[tempi * n + tempj]];
                    tempi = Q[tempi * n + tempj];
                }
                if (abs(temppath - D[i * n + j]) > tolerance) {
                    printf("ERROR 2 in i %d j %d, expected %f and is %f \n", i, j, temppath, D[i * n + j]);
                }
            }
        }
    }
    printf("OK \n");
    printf("Initiating Path Rebuilding Check for GPU DATA.... \n");

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            if ((i == j) && (host_D[i * n + j] != 0)) {
                printf(" ERROR, i=j but element is not 0 \n");
            }
            if (host_D[i * n + j] == inf) {
                if (host_Q[i * n + j] != -2) {
                    printf("ERROR, No path exists between i %d and %d but array Q saysotherwise \n", i, j);
                }
            } else if (host_Q[i * n + j] == -1) {
                if (abs(host_A[i * n + j] - host_D[i * n + j]) > tolerance) {
                    printf("ERROR, from i %d to j %d cheapest route should be the direct one. host_D array not equal to host_A cost \n", i, j);
                }
            } else {
                float temppath = 0;
                int tempi = i;
                int tempj = j;
                while (host_Q[tempi * n + tempj] >= -1) {
                    if (host_Q[tempi * n + tempj] == -1) {
                        temppath = temppath + host_A[tempi * n + tempj];
                        break;
                    }
                    temppath = temppath + host_D[tempi * n + host_Q[tempi * n + tempj]];
                    tempi = host_Q[tempi * n + tempj];
                }
                if (abs(temppath - host_D[i * n + j]) > tolerance) {
                    printf("Path REBUILT ERROR in cheching dinstance from in i %d j %d, expected %f and we found %f \n", i, j, temppath, host_D[i * n + j]);
                }
            }
        }
    }
    printf("OK \n");
    printf("Head to head of shortest path between any two nodes based on array Q from CPU and array Q from blocked GPU algorithm.... \n");

    for (i = 0; i < n; i++) {
        for (j = 0; j < n; j++) {
            //GPU
            float gputemppath = 0;
            int gputempi = i;
            int gputempj = j;
            while (host_Q[gputempi * n + gputempj] >= -1) {
                if (host_Q[gputempi * n + gputempj] == -1) {
                    gputemppath = gputemppath + host_D[gputempi * n + gputempj];
                    break;
                }
                gputemppath = gputemppath + host_D[gputempi * n + host_Q[gputempi * n + gputempj]];
                gputempi = host_Q[gputempi * n + gputempj];
            }
            float cputemppath = 0;
            int cputempi = i;
            int cputempj = j;
            while (Q[cputempi * n + cputempj] >= -1) {
                if (Q[cputempi * n + cputempj] == -1) {
                    cputemppath = cputemppath + A[cputempi * n + cputempj];
                    break;
                }
                cputemppath = cputemppath + D[cputempi * n + Q[cputempi * n + cputempj]];
                cputempi = Q[cputempi * n + cputempj];
            }
            //CPU
            if (abs(gputemppath - cputemppath) > tolerance) {
                printf("ERROR mismatch gpupath is %f cpupath is %f \n", gputemppath, cputemppath);
            } else {
                //printf("WAS OK, VALUES, CPU %f, GPU %f \n",cputemppath,gputemppath);
            }
        }
    }
     */

    printf("ALL OK WE ARE DONE \n");
    return 0;
}

