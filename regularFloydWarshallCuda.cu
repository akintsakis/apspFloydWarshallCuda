#include "hip/hip_runtime.h"
/* Kintsakis Athanasios AEM 6667 */

#define inf 9999
__global__ void funct(int n, int k, float* x, int* qx)
{
	
	int ix= blockIdx.x*blockDim.x + threadIdx.x;

		//Epeksigisi
		/*
		float temp2=x[i*n+k] + x[k*n+j];
		omws
		i=ix/n;		
		
		kai
		j=ix%n = ix&(n-1)
		i*n = ix/n * n = ix-ix%n= ix-j
				
		*/

		
		int j=ix&(n-1);
		float temp2=x[ix-j+k]+x[k*n+j];
		
		if(x[ix]>temp2)
		{
			x[ix]=temp2;
			qx[ix]=k;
		}
		
		if(x[ix]==inf)
		{
			qx[ix]=-2;
		}		
		
}
__global__ void funct2(int n, int k, float* x, int* qx)
{
	
	int ix= blockIdx.x*blockDim.x + threadIdx.x;

		int j=ix&(n-1);
		float temp2=x[ix-j+k]+x[k*n+j];
		if(x[ix]>temp2)
		{
			x[ix]=temp2;
			qx[ix]=k;
		}
		
}


#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include  <time.h>
#include <sys/time.h>


int main(int argc, char **argv)
{

  struct timeval first, second, lapsed,third,fourth,lapsed2;
  struct timezone tzp,tzp2;
	
	
	float *host_A, *host_D;
	
	int *host_Q;
	float *dev_x;
	int *dev_qx;
	float *A;
	int *Q;
	float *D;

	int i,j,bk;
	int k=0;
	
	int n = atoi(argv[1]);
	
	printf("\n");
	printf("RUNNING WITH %d VERTICES \n",n);
	printf("\n");

	hipMalloc(&dev_x, n*n*sizeof(float));
	hipMalloc(&dev_qx, n*n*sizeof(float));
	

    //CPU arrays
	A=(float *)malloc(n*n*sizeof(float));  //arxikos pinakas A
	D=(float *)malloc(n*n*sizeof(float));  //arxikos pinakas D
	Q=(int *)malloc(n*n*sizeof(int));  //arxikos pinakas Q
	
	//GPU arrays
	host_A=(float *)malloc(n*n*sizeof(float));
	host_D=(float *)malloc(n*n*sizeof(float));
	host_Q=(int *)malloc(n*n*sizeof(int));
	


	
	srand(time(NULL));
	
	for(i=0;i<n;i++)
	{
		
		for(j=0;j<n;j++)
		{
			if(i==j)
			{A[i*n+j]=0;}
			else
			{
				A[i*n+j]=1200 * (float) rand()/RAND_MAX + 1;
				if(A[i*n+j]>1000)
				{
					A[i*n+j]=inf;
				}
			}
		}
	}
	
	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		{
			Q[i*n+j]=-1;
		}
	}

	
	for(i=0;i<n;i++)    
	{
		for(j=0;j<n;j++)
		{
			D[i*n+j]=A[i*n+j];
		}

	}

	
	for(i=0;i<n;i++)    //Antigrafi tou A ston host_A
	{
		for(j=0;j<n;j++)
		{
			host_A[i*n+j]=A[i*n+j];
		}

	}


	for(i=0;i<n;i++)    //Antigrafi tou Q ston host_Q
	{
		for(j=0;j<n;j++)
		{
			host_Q[i*n+j]=Q[i*n+j];
		}

	}
gettimeofday(&third, &tzp2);
////////////////////////////First Mem Copy////////////////////

 	gettimeofday(&first, &tzp);
	hipMemcpy(dev_x, host_A, n*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_qx, host_Q, n*n*sizeof(int), hipMemcpyHostToDevice);
	


gettimeofday(&second, &tzp);

  if(first.tv_usec>second.tv_usec){
    second.tv_usec += 1000000;
    second.tv_sec--;
  }
  
  lapsed.tv_usec = second.tv_usec - first.tv_usec;
  lapsed.tv_sec = second.tv_sec - first.tv_sec;

  printf("First Transfer CPU to GPU  Time elapsed: %lu, %lu s\n", lapsed.tv_sec, lapsed.tv_usec);
////////////////////////////////////////////////////GPU Calculation////////////////////////////////
	
	bk=(int)(n*n/512);
	int gputhreads=512;
	
	if(bk>0)
	{
		gputhreads=512;
	}
	else
	{
		bk=1;
		gputhreads=n*n;
	}	

	printf(" \n");
	printf("BLOCKS :   %d      GPU THREADS:     %d \n",bk,gputhreads);
	printf(" \n");	

	
gettimeofday(&first, &tzp);	
	
	funct<<<bk,gputhreads>>>(n,k,dev_x,dev_qx);
	
	for(k=1;k<n;k++)
	{		
		funct2<<<bk,gputhreads>>>(n,k,dev_x,dev_qx);
	}
	
hipDeviceSynchronize();

gettimeofday(&second, &tzp);

  if(first.tv_usec>second.tv_usec){
    second.tv_usec += 1000000;
    second.tv_sec--;
  }
  
  lapsed.tv_usec = second.tv_usec - first.tv_usec;
  lapsed.tv_sec = second.tv_sec - first.tv_sec;

  printf("GPU Calculation Time elapsed: %lu, %lu s\n", lapsed.tv_sec, lapsed.tv_usec);
  printf("\n");

//////////////////////////////////////////////////////////////////////////Second Mem Copy////////////////////
 gettimeofday(&first, &tzp);

hipMemcpy(host_D, dev_x, n*n*sizeof(float), hipMemcpyDeviceToHost);
hipMemcpy(host_Q, dev_qx, n*n*sizeof(int), hipMemcpyDeviceToHost);
gettimeofday(&second, &tzp);

  if(first.tv_usec>second.tv_usec){
    second.tv_usec += 1000000;
    second.tv_sec--;
  }
  
  lapsed.tv_usec = second.tv_usec - first.tv_usec;
  lapsed.tv_sec = second.tv_sec - first.tv_sec;

  printf("Second Transfer GPU to CPU  Time elapsed: %lu, %lu s\n", lapsed.tv_sec, lapsed.tv_usec);
  printf("\n");
//////////////////////////////////////////////////////////////////////

gettimeofday(&fourth, &tzp2); //total time

  if(third.tv_usec>fourth.tv_usec){
    fourth.tv_usec += 1000000;
    fourth.tv_sec--;
  }
  
  lapsed2.tv_usec = fourth.tv_usec - third.tv_usec;
  lapsed2.tv_sec = fourth.tv_sec - third.tv_sec;

  printf("TOTAL GPU + TRANSFERS  Time elapsed: %lu, %lu s\n", lapsed2.tv_sec, lapsed2.tv_usec);







//////////////////////////////////////////////////////////////


//CPU RUN 
printf(" THIS IS CPU \n");
printf("\n");
 gettimeofday(&first, &tzp);

	for(k=0;k<n;k++)
	{
		for(i=0;i<n;i++)
		{
			for(j=0;j<n;j++)
			{

				if((D[i*n+k]+D[k*n+j])<D[i*n+j])
				{
					D[i*n+j]=D[i*n+k]+D[k*n+j];
					Q[i*n+j]=k;					
				}
				if(D[i*n+j]==inf)
				{
					Q[i*n+j]=-2;
				}
			}
		}
	}
/////////////////////////////////////////////////////////////////
gettimeofday(&second, &tzp);

  if(first.tv_usec>second.tv_usec){
    second.tv_usec += 1000000;
    second.tv_sec--;
  }
  
  lapsed.tv_usec = second.tv_usec - first.tv_usec;
  lapsed.tv_sec = second.tv_sec - first.tv_sec;

  printf("CPU Time elapsed: %lu, %lu s\n", lapsed.tv_sec, lapsed.tv_usec);
/////////////////////////////////////////////////////

printf(" \n");

		
printf(" \n");

////////////////////////////////////////////////////////////////////////////////////////


/////////////FROM HERE AND UNDER ARE VALIDATION RUNS

printf("VALIDATING THAT D array from CPU and host_D array from GPU match... \n");


for(i=0;i<n;i++)
{
	for(j=0;j<n;j++)
	{
		if(abs(D[i*n+j]-host_D[i*n+j])>0.0001)
		{
			printf("ERROR MISMATCH in array D i %d j %d \n",i,j);
		}
	}
}
printf("OK \n");
printf("VALIDATING THAT Q array from CPU and host_Q match.... \n");


for(i=0;i<n;i++)
{
	for(j=0;j<n;j++)
	{
		if(Q[i*n+j]!=host_Q[i*n+j])
		{
			printf("ERROR MISMATCH in array Q i %d j %d \n",i,j);
		}
	}
}
printf("OK \n");
//printf("more validating .................... Patience!....\n");

printf("Initiating Path Rebuilding Check for CPU DATA... \n");
for(i=0;i<n;i++)
{
	for(j=0;j<n;j++)
	{

		if( (i==j) && (D[i*n+j]!=0) )
		{
			printf(" ERROR, i=j but element is not 0 \n");
		}

		if(D[i*n+j]==inf)
		{
			if(Q[i*n+j]!=-2)
			{
				printf("ERROR, No path exists between i %d and %d but array Q saysotherwise \n",i,j);
			}
			
		}
		else if(Q[i*n+j]==-1)
		{
			if(abs(A[i*n+j]-D[i*n+j])>0.0001)
			{
				printf("ERROR, from i %d to j %d cheapest route should be the direct one. D array not equal to array A cost \n",i,j);
			}
		}
		else
		{
			float temppath=0;
			int tempi=i;
			int tempj=j;
		
			while(Q[tempi*n+tempj]>=-1)
			{
				if(Q[tempi*n+tempj]==-1)
				{
					temppath=temppath+A[tempi*n+tempj];
					break;
				}
				temppath=temppath+D[tempi*n+Q[tempi*n+tempj]];
				tempi=Q[tempi*n+tempj];
				
			}

			if(abs(temppath-D[i*n+j])>0.0001)
			{
				printf("ERROR 2 in i %d j %d, expected %f and is %f \n",i,j,temppath,D[i*n+j]);
			}
		}
			
				
	}
}
printf("OK \n");
printf("Initiating Path Rebuilding Check for GPU DATA.... \n");

for(i=0;i<n;i++)
{
	for(j=0;j<n;j++)
	{

		if( (i==j) && (host_D[i*n+j]!=0) )
		{
			printf(" ERROR, i=j but element is not 0 \n");
		}

		if(host_D[i*n+j]==inf)
		{
			if(host_Q[i*n+j]!=-2)
			{
				printf("ERROR, No path exists between i %d and %d but array Q saysotherwise \n",i,j);
			}
			
		}
		else if(host_Q[i*n+j]==-1)
		{
			if(abs(host_A[i*n+j]-host_D[i*n+j])>0.0001)
			{
				printf("ERROR, from i %d to j %d cheapest route should be the direct one. host_D array not equal to host_A cost \n",i,j);
			}
		}
		else
		{
			float temppath=0;
			int tempi=i;
			int tempj=j;
		
			while(host_Q[tempi*n+tempj]>=-1)
			{
				if(host_Q[tempi*n+tempj]==-1)
				{
					temppath=temppath+host_A[tempi*n+tempj];
					break;
				}
				temppath=temppath+host_D[tempi*n+host_Q[tempi*n+tempj]];
				tempi=host_Q[tempi*n+tempj];
				
			}
				

			if(abs(temppath-host_D[i*n+j])>0.0001)
			{
				printf("Path REBUILT ERROR in cheching dinstance from in i %d j %d, expected %f and we found %f \n",i,j,temppath,host_D[i*n+j]);
			}
		}
			
				
	}
}
printf("OK \n");
printf("ALL OK WE ARE DONE \n");

return 0;

}
